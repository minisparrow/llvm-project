
#include <hip/hip_runtime.h>

__global__ void vectorAdd(__global__ float *a, __global__ float *b, __global__ float *c) {

  // Get index of current thread
  int i = threadIdx.x;

  // Make sure we do not go out of bounds
  if (i < 16) {
     c[i] = a[i] + b[i];
  }

}
